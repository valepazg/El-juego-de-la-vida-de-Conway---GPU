#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <stdlib.h>
#include <cstdlib>
#include <time.h>
#include <cmath>
#include <fstream>
#include <string>
#include "../include/GameOfLife/grid.hpp"
#include <random>
#include <functional>
#include <sstream> //std::stringstream


using namespace std;
namespace fs = std::filesystem;

class Malla {
private:
  bool* matrix;
  int rows;
  int cols;

public:
  // default constructor
  Malla(){
	this->rows = 10;
	this->cols = 10;
	this->matrix =  new bool[rows*cols];
	std::fill_n(matrix, rows * cols, false);
  };


  Malla(unsigned xin, unsigned yin){
	  if (xin>0 && yin >0){
		  this->rows = xin;
		  this->cols = yin;

		  this->matrix =  new bool[rows*cols];
		  std::fill_n(matrix, rows * cols, false);
	   }
	   else {
		 throw std::invalid_argument("Size of grid, must be rows>0 and cols>0");
	   }
  };

  // square constructor
  Malla(unsigned vin){
  if(vin>0){
	this->rows = vin;
	this->cols = vin;
	this->matrix =  new bool[rows*cols];
	std::fill_n(matrix, rows * cols, false);
	}
  else {
	throw std::invalid_argument("Size of grid, must be vin>0");
  }
};

  // constructor given a string with array
  Malla(string grid){
   vector<vector<bool>> matrix;
   matrix = validate_matrix_str(grid);
   load_grid(matrix); 
  };
  // load grid from file
  Malla(std::filesystem::path path){
  if (std::filesystem::is_regular_file(path)){
	  std::ifstream file(path, std::ios::in | std::ios::binary);
	  if (!file.is_open()){
			throw std::invalid_argument("File cannot be opened");
		  };
	  // Read contents
	  std::string grid{
		std::istreambuf_iterator<char>(file), 
		std::istreambuf_iterator<char>()
	  };
	  // Close the file
	  file.close();
	  vector<vector<bool>> matrix;
	  matrix = validate_matrix_str(grid);
	  load_grid(matrix);
  } else {
	throw std::invalid_argument("File path must be a filename path and be readabale");
  }
  };

  Malla(Malla &old_grid){
  this->rows = old_grid.getRows();
  this->cols = old_grid.getCols();

  this->matrix =  new bool[rows*cols];

  std::fill_n(matrix, rows * cols, false);
 
   for (int i = 0; i < rows; i++)  {
	 for ( int j = 0; j < cols; j++) {
	   this->matrix[i * cols + j] = old_grid.check_next(i,j);
	 }
   }
};
  // check next
  bool check_next(unsigned xin, unsigned yin){
  if (xin<rows && yin<cols) {
	  unsigned check = 0;
	  unsigned up = 0;
	  up = rows-1;
	  if (xin > 0){
		up = xin - 1;
	  } 
	 

	  unsigned down = xin + 1;
	  if (down==rows) {
		down=0;
	  }

	  unsigned left = 0;
	  left = cols -1;
	  if (yin>0){
		left = yin - 1;
	  } 

	  unsigned right = yin + 1;
	  if (right==cols) {
		right=0;
	  }

	  bool cell =  getValue(xin,yin);
	  int up_val = getValue(up,yin);
	  int down_val = getValue(down,yin);
	  int left_val = getValue(xin,left);
	  int right_val = getValue(xin,right);
	  int up_east_val = getValue(up, right);
	  int up_west_val = getValue(up, left);
	  int down_east_val = getValue(down, right);
	  int down_west_val = getValue(down, left);

	  check = up_val + down_val + left_val + right_val+up_east_val+up_west_val+down_east_val+down_west_val;
	  if (cell==false && check==3) {
		return true;
	  } else if (cell==true && (check==2 || check==3)) {
		return true;
	  } else {
		return false;
	  }
  } else {
	throw std::invalid_argument("Índices fuera de rango");
  }
  };
  // load grid
  void load_grid(vector<vector<bool>> &matrix){
    this->rows = matrix.size();
	this->cols = matrix[0].size();
	this->matrix =  new bool[rows*cols];

	for (int i=0;i<rows; i++) {
	for (int j=0;j<cols; j++) {
	  this->matrix[i*cols+j]=matrix[i][j];
	}
	}
  };

  void gen_random(int seed){
	  srand(seed);
	  auto gen = std::bind(std::uniform_int_distribution<>(0,1),std::default_random_engine());

		for (int i=0;i<getRows(); i++) {
		for (int j=0;j<getCols(); j++) {
		  bool random = gen();
		  this->matrix[i*cols + j] = random;
		  }
		}
  };

  // load grid from string
  // generate next grid
  // show grid
  friend std::ostream& operator<<(std::ostream &os, Malla &grid){
   for (int i = 0; i < grid.getRows(); i++)  {
	 for ( int j = 0; j < grid.getCols(); j++) {
	   int value = grid.getValue(i,j);
	   string symbol = value?"X":"O";
	   os << symbol << " ";
	 }
	 os<<endl;
   } 

 return os;

  };

  int getRows(){
	  return this->rows;
  };

  int getCols(){
	  return this->cols;
  };

  bool getValue(int i, int j){
  if (i<getRows() && j<getCols()) {
	return this->matrix[i*cols+j];
  } else {
	throw std::invalid_argument("Indexes must be low than number of rows and cols");
  }
  };

  bool operator==(const Malla& other) const{
	if (rows != other.rows || cols != other.cols){
	  return false;
	}

	for (int i=0;i<rows*cols; i++) {
	  int check = other.matrix[i];
	  if(matrix[i]!=check) {
		return false;
	  }
	}
	return true;
  };

  bool has_life() {
	for (int i=0;i<getRows(); i++) {
	for (int j=0;j<getCols(); j++) {
	  bool val = getValue(i,j);
	  if (val) {
		return true;
	  }
	  }
	}
	return false;
  };

  int alive(){
	int alive = 0;
	for (int i=0;i<getRows(); i++) {
	for (int j=0;j<getCols(); j++) {
	  bool val = getValue(i,j);
	  if (val) {
		alive++;
	  }
	  }
	}
	return alive;
  };
  int dead(){
	int dead = 0;
	for (int i=0;i<getRows(); i++) {
	for (int j=0;j<getCols(); j++) {
	  bool val = getValue(i,j);
	  if (!val) {
		dead++;
	  }
	  }
	}
	return dead;
  };

  ~Malla(){
	  delete[] matrix;
  };

  bool *getMatrix(){
	  return this->matrix;
  }

};

vector<vector<bool>> validate_matrix_str(string grid){

   vector<vector<bool>> matrix;

   std::string line;
   std::string symbol;

   std::string sym_x = "X";
   std::string sym_o = "O";
  
   vector<int> counter;
   vector<vector<string>> matrix_str;
   int row=0;
   int column=0;
   std::istringstream s_grid(grid);
   int count_chars = 0;


   while (std::getline(s_grid, line, '\n')){

	 //std::cerr << line << std::endl;

	 vector<string> newline;
	 count_chars = 0;
	 std::istringstream s_line(line);
	 while (std::getline(s_line, symbol, ',')) {
	   //std::cerr << symbol << std::endl;

	   if (symbol==sym_x || symbol==sym_o) {
		 count_chars++;
		 newline.push_back(symbol);
	   } else {
		  throw std::invalid_argument("Symbol must be 'X' for alive or 'O' for dead separated by comma ','");
	   }
	   column++;
	 }
 	 //std::cerr << "newline ok" << std::endl;

	 matrix_str.push_back(newline);

	 //std::cerr << "matrix pushback" << std::endl;
	 if (counter.size() == 0) {
	   //std::cerr << "count-chars" << count_chars << std::endl;
	   counter.push_back((int)count_chars);
	 }
	 else if (counter.back() == count_chars){ 
	   //std::cerr << "count-chars" << count_chars << std::endl;
 
	   counter.push_back((int)count_chars);
	 } else {
	   throw std::invalid_argument("Line must be same number of symbols");
	 }

	 row++;
	 column=0;
	}
	/**/
   
   int columns = counter.back();
   matrix.resize(row, vector<bool>(columns, false));
   
   for (int i=0;i<row;i++) {
	 for (int j=0;j<columns;j++) {
	   int result =  matrix_str[i][j]==sym_x;
	   matrix[i][j] = result;
	 }
   }

   return matrix;
}

/*
Define kernel
*/

__device__ bool getValue(const bool *matrix, int i, int j, int cols){
	return matrix[i*cols+j];
}

__global__ void gol_next_grid(
						  bool *start, 
						  bool *output, 
						  int cols, 
						  int rows){

  int worldSize = cols * rows;
  for (uint cellId = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
      cellId < worldSize;
      cellId += blockDim.x * gridDim.x) {

 	int xin = cellId*cols;//blockIdx.x * blockDim.x + threadIdx.x;
 	int yin = cellId - xin;//blockIdx.y * blockDim.y + threadIdx.y;

  
	int cell = getValue(start, xin, yin, cols);

	unsigned up = 0;
	up = rows-1;
	if (xin > 0){
	  up = xin - 1;
	} 

	unsigned down = xin + 1;
	if (down==rows) {
	  down=0;
	}

	unsigned left = 0;
	left = cols -1;
	if (yin>0){
	  left = yin - 1;
	} 

	unsigned right = yin + 1;
	if (right==cols) {
	  right=0;
	}

	int up_val = getValue(start, up,yin, cols);
	int down_val = getValue(start, down,yin, cols);
	int left_val = getValue(start, xin,left, cols);
	int right_val = getValue(start, xin,right, cols);
	int up_east_val = getValue(start, up, right, cols);
	int up_west_val = getValue(start, up, left, cols);
	int down_east_val = getValue(start, down, right, cols);
	int down_west_val = getValue(start, down, left, cols);

	unsigned int check = 0;

	if (up_val) {check++;}
	if (down_val) {check++;}
	if (left_val) {check++;} 
	if (right_val) {check++;}
	if (up_east_val) {check++;}
	if (up_west_val) {check++;}
	if (down_east_val) {check++;}
	if (down_west_val) {check++;};

	int position = xin * cols + yin;


	output[position] = (cell==false && check==3) || (cell==true && (check==2 || check==3))?1:0;
	}
}





int main() {
	string filename =  "gpu_cuda_mod1.csv";

	unsigned int m_worldWidth;
	unsigned int m_worldHeight;
	unsigned int m_dataLength = 0;  // m_worldWidth * m_worldHeight

    using std::chrono::high_resolution_clock;
    using std::chrono::duration_cast;
    using std::chrono::duration;
    using std::chrono::milliseconds;

    // abrir un archivo csv para guardar los datos
    ofstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Failed to open file\n";
        return 1;
    }
    // escribir la primera fila con los encabezados de las tablas del .csv
    file << "N, M, Iteration, Cells/sec, Time_sec\n";


	const int SEED = 500;
	
    srand(SEED);

	for (int i=5; i<15;i++){
	/*
	  Generate grid
	 */

		cout << i <<" Grilla de tamano: 2^" << i << "x 2^" << i << endl;

		m_worldWidth = pow(2,i);
		m_worldHeight = pow(2,i);
        m_dataLength = m_worldWidth * m_worldHeight;
		Malla grid(m_worldWidth, m_worldHeight);

		grid.gen_random(SEED);
		if (i<5){
			cout<<grid<<endl;
		}

		bool *hostInput = grid.getMatrix();
		bool *hostOutput = new bool[m_dataLength];

		/*
		  Alojar memoria en device
		 */
		bool *deviceInput;
		bool *deviceOutput;

		hipMalloc((void**)&deviceInput, m_dataLength*sizeof(bool));
		hipMalloc((void**)&deviceOutput, m_dataLength*sizeof(bool));
		
		hipMemcpy(deviceInput, hostInput, m_dataLength*sizeof(bool), hipMemcpyHostToDevice);

		dim3 blockSize(16,16);
		dim3 gridSize((m_worldWidth+blockSize.x-1)/blockSize.x,(m_worldHeight+blockSize.y-1)/blockSize.y);

		/* end init cuda*/

	/*
	  Compute iteration on diferent sizes
	 */	  
		for (int k=0;k<15; k++) {
            auto start_time = high_resolution_clock::now();
	
			// Grid old_grid = grid;
            // Grid grid(old_grid);
			gol_next_grid<<<gridSize,blockSize>>>(
												  deviceInput, 
												  deviceOutput,
												  m_worldWidth,
												  m_worldHeight);

            auto end_time = high_resolution_clock::now();

			hipMemcpy(
					   deviceInput,  
					   deviceOutput, 
					   m_dataLength*sizeof(bool),
					   hipMemcpyDeviceToDevice
					   );
			
        
            // se transforma el resultado a los segundos que tomo la ejecucion
            auto ms_int = duration_cast<milliseconds>(end_time - start_time);
            duration<double, milli> ms_double = end_time - start_time;
            double total_time = ms_double.count();
            double total_time_sec = total_time / 1000;

            // se calcula el numero de celdas evaluadas por segundo
            size_t num_cells_evaluated = m_dataLength;
            double cells_per_second = static_cast<double>(num_cells_evaluated)/static_cast<double>(total_time_sec);


            cout << "Celdas evaluadas por segundo: " << cells_per_second <<" en " << total_time_sec << " segundos. Iteracion: " << k+1 <<  endl;
            // se guardan los datos en un archivo csv
            file << m_worldWidth << "," << m_worldHeight << "," << k << "," << cells_per_second << "," << total_time_sec << "\n";

		}
		//hipFree(deviceInput);
		//hipFree(deviceOutput);

		//delete[]  hostInput;
		//delete[]  hostOutput;


	}


	return 0;
}