#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <stdlib.h>
#include <cstdlib>
#include <time.h>
#include <cmath>
#include <fstream>
#include <string>
#include "../include/GameOfLife/grid.hpp"
#include <random>
#include <functional>
#include <sstream> //std::stringstream


using namespace std;
namespace fs = std::filesystem;

/*
Define kernel
*/

__device__ bool getValue(const bool *matrix, int i, int j, int cols){
	return matrix[i*cols+j];
}



__global__ void gol_next_grid(
						  bool *start, 
						  bool *output, 
						  int cols, 
						  int rows){

  int worldSize = cols * rows;
  for (uint cellId = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
      cellId < worldSize;
      cellId += blockDim.x * gridDim.x) {

 	int xin = cellId*cols;//blockIdx.x * blockDim.x + threadIdx.x;
 	int yin = cellId - xin;//blockIdx.y * blockDim.y + threadIdx.y;

  
	int cell = getValue(start, xin, yin, cols);

	unsigned up = 0;
	up = rows-1;
	if (xin > 0){
	  up = xin - 1;
	} 

	unsigned down = xin + 1;
	if (down==rows) {
	  down=0;
	}

	unsigned left = 0;
	left = cols -1;
	if (yin>0){
	  left = yin - 1;
	} 

	unsigned right = yin + 1;
	if (right==cols) {
	  right=0;
	}

	int up_val = getValue(start, up,yin, cols);
	int down_val = getValue(start, down,yin, cols);
	int left_val = getValue(start, xin,left, cols);
	int right_val = getValue(start, xin,right, cols);
	int up_east_val = getValue(start, up, right, cols);
	int up_west_val = getValue(start, up, left, cols);
	int down_east_val = getValue(start, down, right, cols);
	int down_west_val = getValue(start, down, left, cols);

	unsigned int check = up_val + down_val + left_val + right_val+up_east_val+up_west_val+down_east_val+down_west_val;

	int position = xin * cols + yin;


	output[position] = (cell==false && check==3) || (cell==true && (check==2 || check==3))?1:0;
	}

}





int main() {
	string filename =  "gpu_cuda_mod2.csv";

	unsigned int m_worldWidth;
	unsigned int m_worldHeight;
	unsigned int m_dataLength = 0;  // m_worldWidth * m_worldHeight

    using std::chrono::high_resolution_clock;
    using std::chrono::duration_cast;
    using std::chrono::duration;
    using std::chrono::milliseconds;

    // abrir un archivo csv para guardar los datos
    ofstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Failed to open file\n";
        return 1;
    }
    // escribir la primera fila con los encabezados de las tablas del .csv
    file << "N, M, Iteration, Cells/sec, Time_sec\n";

	srand(500);
	auto gen = std::bind(std::uniform_int_distribution<>(0,1),std::default_random_engine());


	const int SEED = 500;
	
    srand(SEED);

	for (int i=5; i<15;i++){
	/*
	  Generate grid
	 */

		cout << i <<" Grilla de tamano: 2^" << i << "x 2^" << i << endl;

		m_worldWidth = pow(2,i);
		m_worldHeight = pow(2,i);
        m_dataLength = m_worldWidth * m_worldHeight;

        vector<vector<bool>> h_lifeData(m_worldHeight, vector<bool>(m_worldWidth));               
        bool* h_tempLifeData= new bool[m_dataLength];               
        vector<vector<bool>> h_resultLifeData(m_worldHeight, vector<bool>(m_worldWidth));                     	

        for (int j = 0; j < m_worldHeight; j++) {
            for(int l = 0; l < m_worldWidth; l++){
                h_lifeData[j][l] = gen();
                h_tempLifeData[i * m_worldWidth + j] = h_lifeData[j][l];
            }
        }


		/*
		  Alojar memoria en device
		 */
		bool *deviceInput;
		bool *deviceOutput;
		
		// crear espacio de memoria para la matriz
		hipMalloc((void**)&deviceInput, m_dataLength*sizeof(bool));
		hipMalloc((void**)&deviceOutput, m_dataLength*sizeof(bool));
		

		// cpu -> GPU
		hipMemcpy(
				   deviceInput, 
				   h_tempLifeData, 
				   m_dataLength*sizeof(bool), 
				   hipMemcpyHostToDevice);


		//
		dim3 blockSize(16,16);
		dim3 gridSize(
					  (m_worldWidth+blockSize.x-1)/blockSize.x,
					  (m_worldHeight+blockSize.y-1)/blockSize.y);
		/* end init cuda*/

		/*
		  Compute iteration on diferent sizes
		*/	  
		for (int k=0;k<15; k++) {
            auto start_time = high_resolution_clock::now();
	
			// Grid old_grid = grid;
            // Grid grid(old_grid);
			gol_next_grid<<<gridSize,blockSize>>>(
												  deviceInput, 
												  deviceOutput,
												  m_worldWidth,
												  m_worldHeight);

            auto end_time = high_resolution_clock::now();

			hipMemcpy(
					   deviceInput,  
					   deviceOutput, 
					   m_dataLength*sizeof(bool),
					   hipMemcpyDeviceToDevice
					   );
			
        
            // se transforma el resultado a los segundos que tomo la ejecucion
            auto ms_int = duration_cast<milliseconds>(end_time - start_time);
            duration<double, milli> ms_double = end_time - start_time;
            double total_time = ms_double.count();
            double total_time_sec = total_time / 1000;

            // se calcula el numero de celdas evaluadas por segundo
            size_t num_cells_evaluated = m_dataLength;
            double cells_per_second = static_cast<double>(num_cells_evaluated)/static_cast<double>(total_time_sec);


            cout << "Celdas evaluadas por segundo: " << cells_per_second <<" en " << total_time_sec << " segundos. Iteracion: " << k+1 <<  endl;
            // se guardan los datos en un archivo csv
            file << m_worldWidth << "," << m_worldHeight << "," << k << "," << cells_per_second << "," << total_time_sec << "\n";

		}
		//hipFree(deviceInput);
		//hipFree(deviceOutput);

		//delete[]  hostInput;
		//delete[]  hostOutput;


	}


	return 0;
}