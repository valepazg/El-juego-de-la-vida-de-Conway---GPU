#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <stdlib.h>
#include <cstdlib>
#include <time.h>
#include <cmath>
#include <fstream>
#include <string>
#include "../include/GameOfLife/grid.hpp"
#include <random>
#include <functional>
#include <sstream> //std::stringstream

using namespace std;
namespace fs = std::filesystem;
typedef unsigned int uint;

// __global__ void gol_next_grid(int *start, int *output, int cols, int rows) {
// 	int x = blockIdx.x;
// 	int y = blockIdx.y;

// 	output[x][y] = start[x][y]*start[x][y];
// }



int main() {
    using std::chrono::high_resolution_clock;
    using std::chrono::duration_cast;
    using std::chrono::duration;
    using std::chrono::milliseconds;

	const int SIZE = 100;
	unsigned int worldWidth=SIZE;
	unsigned int worldHeight=SIZE;
	unsigned int dataLength = worldHeight * worldWidth; 
    vector<vector<uint>> matrix(worldHeight, vector<uint>(worldWidth));               
	
	for (int j=0;j<worldHeight;j++){
		for (int k=0;k<worldHeight;k++){
			matrix[j][k] = uint(rand() % 10+1);
		}		
	}
	for (int j=0;j<worldHeight;j++){
		for (int k=0;k<worldHeight;k++){
			std::cout<<matrix[j][k]<<" ";
		}		
		std::cout<<std::endl;
	}
	


	int *deviceInput;
	int *deviceOutput;

	hipMalloc((void**)&deviceInput, dataLength*sizeof(bool));
	hipMalloc((void**)&deviceOutput, dataLength*sizeof(bool));

	// hipMemcpy(
	// 		   deviceInput, 
	// 		   matrix, 
	// 		   dataLength*sizeof(bool), 
	// 		   hipMemcpyHostToDevice);

	// dim3 blockSize(16,16);
	// dim3 gridSize((worldWidth+blockSize.x-1)/blockSize.x,
	// 			  (worldHeight+blockSize.y-1)/blockSize.y);


	return 0;
}